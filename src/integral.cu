#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "integral.h"

__constant__ float PI;

__device__ void complexDiv(float *num_r, float *num_i,
                           float *dem_r, float *dem_i,
                           float *res_r, float *res_i) {
   *res_r = ((*num_r)*(*dem_r)+(*num_i)*(*dem_i)) / ((*dem_r)*(*dem_r)+(*dem_i)*(*dem_i));
   *res_i = ((*num_i)*(*dem_r)-(*num_r)*(*dem_i)) / ((*dem_r)*(*dem_r)+(*dem_i)*(*dem_i));
}


__device__ void gpuEval(float *coefficients, int size, float freq,
                        float *real, float *image) {
   float omega = 2.0 * PI * freq ;
   *image = 0.0;
   *real = 0.0 ;

   int maxOrder = size;
   float s = 1 ;

   for(int order = 0 ; order < maxOrder ; ++ order) {
      // i^0 = 1, i^1 = i, i^2 = -1, i^3 = -i
      switch(order & 3) {
         case 0:
            *real  += s * coefficients[order] ; break ;
         case 1:
            *image += s * coefficients[order] ; break ;
         case 2:
            *real  -= s * coefficients[order] ; break ;
         case 3:
            *image -= s * coefficients[order] ; break ;
      }
      s *= omega ;
   }
}

__global__ void freqKernel(float *freq, float *real, float *image,
                           float *tf_num, int tf_num_size,
                           float *tf_den, int tf_den_size,
                           float start, float ratio) {
   int index = blockIdx.x;
   //printf("GPU: %d\n", index);
   float cur_freq = start * powf(ratio, index);
   freq[index] = cur_freq;
   float num_r, num_i, den_r, den_i;
   gpuEval(tf_num, tf_num_size, cur_freq, &num_r, &num_i);
   gpuEval(tf_den, tf_den_size, cur_freq, &den_r, &den_i);
   complexDiv(&num_r, &num_i, &den_r, &den_i, &real[index], &image[index]);
}

void freqGpuSimulate(float *freq, float *real, float *image, float *tf_num, int tf_num_s, float *tf_den, int tf_den_s, float start, float ratio, int kernels) {
  float host_PI = acos(-1.0);

  float *freq_c, *real_c, *image_c, *tf_num_c, *tf_den_c;
  hipMemcpyToSymbol(HIP_SYMBOL("PI"), &host_PI, sizeof(host_PI));
  hipMalloc((void**) &freq_c,  kernels * sizeof(float));
  hipMalloc((void**) &real_c,  kernels * sizeof(float));
  hipMalloc((void**) &image_c, kernels * sizeof(float));
  hipMalloc((void**) &tf_num_c, tf_num_s * sizeof(float));
  hipMalloc((void**) &tf_den_c, tf_den_s * sizeof(float));

  hipMemcpy(tf_num_c, tf_num, tf_num_s * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(tf_den_c, tf_den, tf_den_s * sizeof(float), hipMemcpyHostToDevice);
  freqKernel <<< kernels, 1 >>> (freq_c, real_c, image_c, tf_num_c, tf_num_s, tf_den_c, tf_den_s, start, ratio);
  hipMemcpy(freq, freq_c, kernels * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(real, real_c, kernels * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(image, image_c, kernels * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(freq_c);
  hipFree(real_c);
  hipFree(image_c);
  hipFree(tf_num_c);
  hipFree(tf_den_c);
}

__global__ void gpuIntegral(float *times, float *tf_num, int tf_num_s, float *tf_den, int tf_den_s, float *result) {
  int t = blockIdx.x + 1;
  float lower, upper;
  float num_r, num_i, den_r, den_i, dummy_i;
  gpuEval(tf_num, tf_num_s, times[t-1], &num_r, &num_i);
  gpuEval(tf_den, tf_den_s, times[t-1], &den_r, &den_i);
  complexDiv(&num_r, &num_i, &den_r, &den_i, &lower, &dummy_i);
  gpuEval(tf_num, tf_num_s, times[t], &num_r, &num_i);
  gpuEval(tf_den, tf_den_s, times[t], &den_r, &den_i);
  complexDiv(&num_r, &num_i, &den_r, &den_i, &upper, &dummy_i);
  
  result[t] = (lower + upper) * (times[t] - times[t-1]) / 2;
}

void timeGpuSimulate(float *times, int times_s, float *tf_num, int tf_num_s, float *tf_den,
                     int tf_den_s, float *result) {
  float *result_c, *tf_num_c, *tf_den_c, *times_c;
  hipMalloc((void**) &result_c, times_s * sizeof(float));
  hipMalloc((void**) &times_c, times_s * sizeof(float));
  hipMalloc((void**) &tf_num_c, tf_num_s * sizeof(float));
  hipMalloc((void**) &tf_den_c, tf_den_s * sizeof(float));
  
  hipMemcpy(times_c, times, times_s * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(tf_num_c, tf_num, tf_num_s * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(tf_den_c, tf_den, tf_den_s * sizeof(float), hipMemcpyHostToDevice);
  gpuIntegral <<< (times_s-1), 1 >>> (times_c, tf_num_c, tf_num_s, tf_den_c, tf_den_s, result_c);
  hipMemcpy(result, result_c, times_s * sizeof(float), hipMemcpyDeviceToHost);
  
  result[0] = 0.0;
  for(int i = 1; i < times_s; ++i)
    result[i] += result[i-1];

  hipFree(result_c);
  hipFree(times_c);
  hipFree(tf_num_c);
  hipFree(tf_den_c);
}
