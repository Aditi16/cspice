#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "integral.h"

__constant__ float PI;

__device__ void complexDiv(float *num_r, float *num_i,
                           float *dem_r, float *dem_i,
                           float *res_r, float *res_i) {
   *res_r = ((*num_r)*(*dem_r)+(*num_i)*(*dem_i)) / ((*dem_r)*(*dem_r)+(*dem_i)*(*dem_i));
   *res_i = ((*num_i)*(*dem_r)-(*num_r)*(*dem_i)) / ((*dem_r)*(*dem_r)+(*dem_i)*(*dem_i));
}


__device__ void gpuEval(float *coefficients, int size, float freq,
                        float *real, float *image) {
   float omega = 2.0 * PI * freq ;
   *image = 0.0;
   *real = 0.0 ;

   int maxOrder = size;
   float s = 1 ;

   for(int order = 0 ; order < maxOrder ; ++ order) {
      // i^0 = 1, i^1 = i, i^2 = -1, i^3 = -i
      switch(order & 3) {
         case 0:
            *real  += s * coefficients[order] ; break ;
         case 1:
            *image += s * coefficients[order] ; break ;
         case 2:
            *real  -= s * coefficients[order] ; break ;
         case 3:
            *image -= s * coefficients[order] ; break ;
      }
      s *= omega ;
   }
}

__global__ void freqKernel(float *freq, float *real, float *image,
                           float *tf_num, int tf_num_size,
                           float *tf_den, int tf_den_size,
                           float start, float ratio) {
   int index = blockIdx.x;
   //printf("GPU: %d\n", index);
   float cur_freq = start * powf(ratio, index);
   freq[index] = cur_freq;
   float num_r, num_i, den_r, den_i;
   gpuEval(tf_num, tf_num_size, cur_freq, &num_r, &num_i);
   gpuEval(tf_den, tf_den_size, cur_freq, &den_r, &den_i);
   complexDiv(&num_r, &num_i, &den_r, &den_i, &real[index], &image[index]);
}

void freqGpuSimulate(float *freq, float *real, float *image, float *tf_num, int tf_num_s, float *tf_den, int tf_den_s, float start, float ratio, int kernels) {
  float host_PI = acos(-1.0);

  float *freq_c, *real_c, *image_c, *tf_num_c, *tf_den_c;
  hipMemcpyToSymbol(HIP_SYMBOL("PI"), &host_PI, sizeof(host_PI));
  hipMalloc((void**) &freq_c,  kernels * sizeof(float));
  hipMalloc((void**) &real_c,  kernels * sizeof(float));
  hipMalloc((void**) &image_c, kernels * sizeof(float));
  hipMalloc((void**) &tf_num_c, tf_num_s * sizeof(float));
  hipMalloc((void**) &tf_den_c, tf_den_s * sizeof(float));

  hipMemcpy(tf_num_c, tf_num, tf_num_s * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(tf_den_c, tf_den, tf_den_s * sizeof(float), hipMemcpyHostToDevice);
  freqKernel <<< kernels, 1 >>> (freq_c, real_c, image_c, tf_num_c, tf_num_s, tf_den_c, tf_den_s, start, ratio);
  hipMemcpy(freq, freq_c, kernels * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(real, real_c, kernels * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(image, image_c, kernels * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(freq_c);
  hipFree(real_c);
  hipFree(image_c);
  hipFree(tf_num_c);
  hipFree(tf_den_c);
}

