#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "integral.h"

__constant__ double PI;

__device__ void complexDiv(double *num_r, double *num_i,
                           double *dem_r, double *dem_i,
                           double *res_r, double *res_i) {
   *res_r = ((*num_r)*(*dem_r)+(*num_i)*(*dem_i)) / ((*dem_r)*(*dem_r)+(*dem_i)*(*dem_i));
   *res_i = ((*num_i)*(*dem_r)-(*num_r)*(*dem_i)) / ((*dem_r)*(*dem_r)+(*dem_i)*(*dem_i));
}


__device__ void gpuEval(double *coefficients, int size, double freq,
                        double *real, double *image) {
   double omega = 2.0 * PI * freq ;
   *image = 0.0;
   *real = 0.0 ;

   int maxOrder = size;
   double s = 1 ;

   for(int order = 0 ; order < maxOrder ; ++ order) {
      // i^0 = 1, i^1 = i, i^2 = -1, i^3 = -i
      switch(order & 3) {
         case 0:
            *real  += s * coefficients[order] ; break ;
         case 1:
            *image += s * coefficients[order] ; break ;
         case 2:
            *real  -= s * coefficients[order] ; break ;
         case 3:
            *image -= s * coefficients[order] ; break ;
      }
      s *= omega ;
   }
}

__global__ void freqKernel(double *freq, double *real, double *image,
                           double *tf_num, int tf_num_size,
                           double *tf_den, int tf_den_size,
                           double start, double ratio) {
   int index = blockIdx.x;
   //printf("GPU: %d\n", index);
   double cur_freq = start * powf(ratio, index);
   freq[index] = cur_freq;
   double num_r, num_i, den_r, den_i;
   gpuEval(tf_num, tf_num_size, cur_freq, &num_r, &num_i);
   gpuEval(tf_den, tf_den_size, cur_freq, &den_r, &den_i);
   complexDiv(&num_r, &num_i, &den_r, &den_i, &real[index], &image[index]);
}

void freqGpuSimulate(double *freq, double *real, double *image, double *tf_num, int tf_num_s, double *tf_den, int tf_den_s, double start, double ratio, int kernels) {
  double host_PI = acos(-1.0);

  double *freq_c, *real_c, *image_c, *tf_num_c, *tf_den_c;
  hipMemcpyToSymbol(HIP_SYMBOL("PI"), &host_PI, sizeof(host_PI));
  hipMalloc((void**) &freq_c,  kernels * sizeof(double));
  hipMalloc((void**) &real_c,  kernels * sizeof(double));
  hipMalloc((void**) &image_c, kernels * sizeof(double));
  hipMalloc((void**) &tf_num_c, tf_num_s * sizeof(double));
  hipMalloc((void**) &tf_den_c, tf_den_s * sizeof(double));

  hipMemcpy(tf_num_c, tf_num, tf_num_s * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(tf_den_c, tf_den, tf_den_s * sizeof(double), hipMemcpyHostToDevice);
  freqKernel <<< kernels, 1 >>> (freq_c, real_c, image_c, tf_num_c, tf_num_s, tf_den_c, tf_den_s, start, ratio);
  hipMemcpy(freq, freq_c, kernels * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(real, real_c, kernels * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(image, image_c, kernels * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(freq_c);
  hipFree(real_c);
  hipFree(image_c);
  hipFree(tf_num_c);
  hipFree(tf_den_c);
}

