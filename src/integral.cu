#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_runtime.h>

#include "integral.h"

__constant__ double PI;

__device__ void complexDiv(double *num_r, double *num_i,
                           double *dem_r, double *dem_i,
                           double *res_r, double *res_i) {
   *res_r = ((*num_r)*(*dem_r)+(*num_i)*(*dem_i)) / ((*dem_r)*(*dem_r)+(*dem_i)*(*dem_i));
   *res_i = ((*num_i)*(*dem_r)-(*num_r)*(*dem_i)) / ((*dem_r)*(*dem_r)+(*dem_i)*(*dem_i));
}


__device__ void gpuEval(double *coefficients, int size, double freq,
                        double *real, double *image) {
   double omega = 2.0 * PI * freq ;
   *image = 0.0;
   *real = 0.0 ;

   int maxOrder = size;
   double s = 1 ;

   for(int order = 0 ; order < maxOrder ; ++ order) {
      // i^0 = 1, i^1 = i, i^2 = -1, i^3 = -i
      switch(order & 3) {
         case 0:
            *real  += s * coefficients[order] ; break ;
         case 1:
            *image += s * coefficients[order] ; break ;
         case 2:
            *real  -= s * coefficients[order] ; break ;
         case 3:
            *image -= s * coefficients[order] ; break ;
      }
      s *= omega ;
   }
}

__global__ void freqKernel(double *freq, double *real, double *image,
                           double *tf_num, int tf_num_size,
                           double *tf_den, int tf_den_size,
                           double start, double ratio) {
   int index = blockIdx.x;
   double cur_freq = start * powf(ratio, index);
   freq[index] = (double)index;
   //freq[index] = cur_freq;
   double num_r, num_i, den_r, den_i;
   gpuEval(tf_num, tf_num_size, cur_freq, &num_r, &num_i);
   gpuEval(tf_den, tf_den_size, cur_freq, &den_r, &den_i);
   complexDiv(&num_r, &num_i, &den_r, &den_i, &real[index], &image[index]);
}

void freqGpuSimulate(SimulateConfig &config, Simulator::TransferFunction &tf, vector<pair<double,complex<double> > > &result) {
  double ratio = exp(log(10.0) / config.step);
  int kernels = (int)(log(config.end / config.start) / log(ratio));
  printf("start: %lf, end: %lf, step: %lf\n", config.start, config.end, config.step);
  printf("ratio: %lf, kernels: %d\n", ratio, kernels);
  double host_PI = acos(-1.0);

  double *freq, *real, *image, *tf_num, *tf_den;
  freq = (double*)malloc(kernels * sizeof(double));
  real = (double*)malloc(kernels * sizeof(double));
  image = (double*)malloc(kernels * sizeof(double));
  tf_num = (double*)malloc(tf.num.size() * sizeof(double));
  tf_den = (double*)malloc(tf.den.size() * sizeof(double));

  for(int i = 0; i < tf.num.size(); ++i)
    tf_num[i] = tf.num[i];
  for(int i = 0; i < tf.den.size(); ++i)
    tf_den[i] = tf.den[i];

  double *freq_c, *real_c, *image_c, *tf_num_c, *tf_den_c;
  hipMemcpyToSymbol(HIP_SYMBOL("PI"), &host_PI, sizeof(host_PI));
  hipMalloc((void**) &freq_c,  kernels * sizeof(double));
  hipMalloc((void**) &real_c,  kernels * sizeof(double));
  hipMalloc((void**) &image_c, kernels * sizeof(double));
  hipMalloc((void**) &tf_num_c, tf.num.size() * sizeof(double));
  hipMalloc((void**) &tf_den_c, tf.den.size() * sizeof(double));

  hipMemcpy(tf_num_c, tf_num, tf.num.size() * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(tf_den_c, tf_den, tf.den.size() * sizeof(double), hipMemcpyHostToDevice);
  freqKernel <<< kernels, 1 >>> (freq_c, real_c, image_c, tf_num_c, tf.num.size(), tf_den_c, tf.den.size(), config.start, ratio);
  hipMemcpy(freq, freq_c, kernels * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(real, real_c, kernels * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(image, image_c, kernels * sizeof(double), hipMemcpyDeviceToHost);

  for(int i = 0; i < kernels; ++i) {
    printf("%d: %lf\n", i, freq[i]);
    result.push_back(pair<double , complex<double> >(freq[i], complex<double>(real[i], image[i]))) ;
  }
  hipFree(freq_c);
  hipFree(real_c);
  hipFree(image_c);
  hipFree(tf_num_c);
  hipFree(tf_den_c);

  free(freq);
  free(real);
  free(image);
  free(tf_num);
  free(tf_den);
}

